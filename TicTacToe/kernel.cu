#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h> 
#include <time.h>
#include <sys/utime.h>
struct Simulation {
	int *board;
	int *win;
	int *rows;
	int *columns;
	int *possible;
};

__device__
void checkWin(Simulation *sim)
{
	sim->win[0] = 0;
	for (int j = 0; j < (sim->columns[0]); j++)
	{
		if (sim->board[0*sim->columns[0] + j] == sim->board[1*sim->columns[0] + j] 
			&& sim->board[0*sim->columns[0] + j] == sim->board[2*sim->columns[0] + j] && sim->board[0*sim->columns[0] + j] != 0)
		{
			sim->win[0] = sim->board[0*sim->columns[0] + j];
		}
	}
	for (int j = 0; j < (sim->rows[0]); j++)
	{
		if (sim->board[j*sim->columns[0] + 0] == sim->board[j*sim->columns[0] + 1] 
			&& sim->board[j*sim->columns[0] + 0] == sim->board[j*sim->columns[0] + 2] && sim->board[0 * sim->columns[0] + j] != 0)
		{
			sim->win[0] = sim->board[j*sim->columns[0] + 0];
		}
	}
	if (sim->board[0] == sim->board[4] && sim->board[0] == sim->board[8] && sim->board[0] != 0)
	{
		sim->win[0] = sim->board[0];
	}
	if (sim->board[2] == sim->board[4] && sim->board[2] == sim->board[6] && sim->board[2] != 0)
	{
		sim->win[0] = sim->board[2];
	}
}


void printBoard(Simulation sim)
{
	printf("columns[0]: %i", sim.columns[0]);
	printf("\nrows[0]: %i\n", sim.rows[0]);
	for (int i = 0; i < sim.rows[0]; i++)
	{
		for (int j = 0; j < sim.columns[0]; j++)
		{
			printf("  %i  ", sim.board[i * sim.columns[0] + j]);
		}
		printf("\n");
	}
	printf("\n");
}

void createSim(Simulation *sim)
{
	hipMallocManaged((void **)&sim->rows, sizeof(int));
	hipMallocManaged((void **)&sim->columns, sizeof(int));
	sim->rows[0] = 6;
	sim->columns[0] = 7;
	hipMallocManaged((void **)&sim->board, sizeof(int) * sim->columns[0] * sim->rows[0]);
	for (int i = 0; i < sim->columns[0] * sim->rows[0]; i++)
	{
		sim->board[i] = 0;
	}
	hipMallocManaged((void **)&sim->possible, sizeof(int) * 8);
	hipMallocManaged((void **)&sim->win, sizeof(int));
	sim->win[0] = 0;
	
	
}


__device__
void possibleMoves(Simulation *sim)
{
	int moves = 0;
	for (int i = 0; i < sim->columns[0]; i++)
	{
		for (int j = 0; j < sim->rows[0]; j++)
		{
			if (sim->board[j*sim->columns[0] + i] == 0)
			{
				moves++;
				sim->possible[moves] = j*sim->columns[0] + i;
			}
		}
	}
	sim->possible[0] = moves;
}

void resetBoard(Simulation *sim)
{
	//printf("A");
	//printf("\nrows[0]: %i, columns[0]: %i", sim.rows[0], sim.columns[0]);
	for (int i = 0; i < sim->columns[0] * sim->rows[0]; i++)
	{
		//printf("B");
		sim->board[i] = 0;
	}
	//printf("C");
	sim->win[0] = 0;
	sim->possible[0] = 1;
	//printf("D");
}

__device__
int randomMove(Simulation *sim, int player, unsigned int seed)
{
	float*r;
	int threads = 1;
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,
		HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen,
		1234ULL);
	hiprandGenerateUniform(gen, r, 1);
	int val = 0;
	if (sim->possible[0] > 0)
	{
		int rInt = r[0] * sim->possible[0];
		val = sim->possible[rInt + 1];
		sim->board[val] = player;

	}
	else
	{
		val = 0;
	}
	
	/*hiprandState_t states;
	//printf("\nSeed Setter: %i", threadIdx.x + blockIdx.x);
	hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
	//	threadIdx.x + blockIdx.x, /* the sequence number should be different for each core (unless you want all
	//							  cores to get the same sequence of numbers for some reason - use thread id! */
	//	0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
	//	&states);/* [blockIdx.x * threads + threadIdx.x]);*/
	/*int val = 0;
	if (sim->possible[0] > 0)
	{
		int r = hiprand(&states) % sim->possible[0];
		val = sim->possible[r + 1];
		sim->board[val] = player;

	}
	else
	{
		val = 0;
	}
	*/
	//free(&states);
	return val;

	//printf("The board value: %i\n", val);
}

__global__
void handleComputer(Simulation **sim, int seed)
{
	//printf("A");
	int player = -1;
	//printf("B");
	possibleMoves(sim[0]);
	//printf("C");
	randomMove(sim[0], player, seed);
	//printf("D");
}

__device__
int resultOfMove(Simulation *sim, int seed)
{
	int move = -1;
	int player = 1;
	if (sim->possible[0] != 0)
	{
		move = randomMove(sim, player, seed);
		if (player == 1) { player = -1; }
		else { player = 1; }
		possibleMoves(sim);
		//printf("9");
		checkWin(sim);
	}
	while (sim->win[0] == 0 && sim->possible[0] != 0)
	{
		seed++;
		//printf("\n7 + INDEX: %i", blockIdx.x * blockDim.x + threadIdx.x);
		randomMove(sim, player, seed);
		//printf("\n8 + INDEX: %i", blockIdx.x * blockDim.x + threadIdx.x);
		if (player == 1) { player = -1; }
		else { player = 1; }
		//printf("\n9 + INDEX: %i", blockIdx.x * blockDim.x + threadIdx.x);
		possibleMoves(sim);
		//printf("\n10 + INDEX: %i", blockIdx.x * blockDim.x + threadIdx.x);
		checkWin(sim);
		//printf("\n11 + INDEX: %i", blockIdx.x * blockDim.x + threadIdx.x);
	}
	//printf("\nDONE");
	return move;
}
__global__
void computerMove(Simulation **sim, int runs, int blocks, int threads, unsigned int seed, int *move)
{

	//printf("2");
	//printf("\nsim[0] rows[0]: %i\n", sim[0]->rows[0]);
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	/*printf("\nStride: %i", stride);
	printf("\nGridDim: %i", gridDim.x);
	printf("\nBlockDim: %i", blockDim.x);
	printf("\nIndex: %i", index);*/
	//printf("3");
	for (int i = index; i < runs; i += stride)
	{


		//seed++;
		//printf("5");
		possibleMoves(sim[i]);
		//printf("\n6 + INDEX: %i", index);
		int location = resultOfMove(sim[i], seed);
		//printf("\nIndex: %i\nLOCATION: %i", index, location);
		move[location] += sim[i]->win[0];
	}
	//printf("A");
}

__global__
void loopStarter(Simulation **sim)
{
	possibleMoves(sim[0]);
	checkWin(sim[0]);
}

int main(void)
{
	const int runs = 4096;
	int blockSize = 256;
	int numBlocks = (runs + blockSize - 1) / blockSize;
	// Allocate Unified Memory -- accessible from CPU or GPU
	Simulation **sim;// = new Simulation*();
					 //createSim(sim);
	hipMallocManaged(&sim, sizeof(Simulation)*runs);
	for (int i = 0; i < runs; i++)
	{

		sim[i] = new Simulation();
		hipMallocManaged(&sim[i], sizeof(Simulation));
		createSim(sim[i]);

	}
	sim[0]->possible[0] = 1;
	int wins = 0;
	int ties = 0;
	int losses = 0;
	for (int i = 0; i < 1000; i++)
	{
		while (sim[0]->win[0] == 0 && sim[0]->possible[0] > 0)
		{
			int *board;
			hipMallocManaged(&board, sizeof(int) * sim[0]->columns[0] * sim[0]->rows[0]);
			for (int i = 0; i < sim[0]->columns[0] * sim[0]->rows[0]; i++)
			{
				board[i] = sim[0]->board[i];
			}
			int *move;// = new int[sim->columns[0] * sim->rows[0]];
			hipMallocManaged(&move, sizeof(int) * sim[0]->columns[0] * sim[0]->rows[0]);
			for (int i = 0; i < sim[0]->columns[0] * sim[0]->rows[0]; i++)
			{
				move[i] = 0;
			}
			srand(time(NULL));
			//printf("Start");

			computerMove << <numBlocks, blockSize >> > (sim, runs, numBlocks, blockSize, rand(), move);

			hipDeviceSynchronize();



			int maxValue = -1;
			int max = -1;
			for (int i = 0; i < runs; i++)
			{
				if (move[i] > maxValue)
				{
					maxValue = move[i];
					max = i;
				}
			}
			board[max] = 1;

			sim[0]->board = board;
			//printBoard(*sim[0]);
			handleComputer << <1, 1 >> > (sim, rand());
			hipDeviceSynchronize();
			//printBoard(*sim[0]);
			for (int i = 1; i < runs; i++)
			{
				sim[i]->board = sim[0]->board;
			}
			loopStarter << <1, 1 >> > (sim);
			hipDeviceSynchronize();
			//printBoard(*sim[0]);
			hipFree(&board);
			hipFree(&move);
			//printf("\nPossible Moves: %i and Win?: %i\n", sim[0]->possible[0], sim[0]->win[0]);
		}
		if (sim[0]->win[0] == 1)
		{
			wins++;
		}
		else if (sim[0]->win[0] == 0)
		{
			ties++;
		}
		else
		{
			losses++;
		}

		for (int i = 0; i < runs; i++)
		{
			resetBoard(sim[i]);
		}
		printf("\nITERATION: %i, TOTAL WINS: %i  Ties: %i  Losses: %i", i, wins, ties, losses);
	}
	printf("\nWins By Player One: %i  Ties: %i  Losses: %i... of %i total games.", wins, ties, losses, 1000);

	// Run kernel on 1M elements on the CPU



	//hipFree(&sim);

	return 0;
}